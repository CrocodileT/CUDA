#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/*
Идея решения, записать все возможные варианты и выбрать из них лучщший
*/

//Анализирует все варианты проходов и записывает их в stat_way и stat_call
__global__ void AllCall(char* str, int* size_str, int* stop_call, int *stat_way, int *stat_call) {
	//Получаем id текущего треда.
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int i = 1;
	stat_way[idx]= 0;
	stat_call[idx] = 1;
	int offset = 0;
	if (idx > 0) {
		for (int j = 0; j < idx; j++) {
			offset += size_str[j];
		}
	}

	while (str[offset + i] != '\n') {
		if (str[offset + i] != str[offset + i - 1]) {
			stat_call[idx] += 1;
			if (stat_call[idx] > *stop_call) {
				stat_way[idx] = i;
				return;
			}
		}
		i++;
	}
	stat_way[idx] = i;
	return;
}

//Эти две функции пробегают по статистике и выбирают лучший вариант так, чтобы он отвечал условию на максимальное количество проходов
//Рекурсивно пробегает по статистике
__device__ void recurs(int* analys_way, int* analys_call, int stop_call, int leng_data_size, int call, int way, int idx, int *res) {
	if (call >= stop_call) {
		if (way > res[0]) {
			res[0] = way;
		}
		return;
	}
	else {
		for (int i = idx; i < leng_data_size; i++) {
			recurs(analys_way, analys_call, stop_call, leng_data_size, call + analys_call[i], way + analys_way[i], idx+1, res);
		}
		if (way > res[0]) {
			res[0] = way;
		}
	}
	return;
}

//Вызывает рекурсию
//Нельзя сразу вызвать рекурсию GPU с CPU, для этого нужна эта функция
__global__ void Analys(int* analys_way, int* analys_call, int* res_analys, int* stop_call, int * leng_data_size) {
	//Получаем id текущего треда.
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int *res = (int*)malloc(sizeof(int));
	res[0] = 0;
	recurs(analys_way, analys_call, *stop_call, *leng_data_size, 0, 0, idx, res);
	printf("%d\n", res[0]);
	res_analys[idx] = res[0];
	return;
}


__host__ int main() {
	FILE *file; //файл
	int stop_call; //Количество проходов после которых устает
	int all_sum = 0; //Количество всех элементов в файле
	int *data_size = (int*)malloc(sizeof(int)); //Массив длин строк сообщение
	char name[20]; //Имя файла
	char elem; //Сюда записываем по элементу из файла
	char *all_data = (char*)malloc(sizeof(char)); //Тут храним все сообщение

	//Получаем имя файла и усталость
	printf("Enter name fail: ");
	gets_s(name);
	printf("Enter name tired: ");
	scanf_s("%d", &stop_call);

	//проверяем открывается ли файл
	if ((file = fopen(name, "r")) == false) {
		printf("Error open fail");
		system("pause");
		return 0;
	}

	elem = fgetc(file);

	//Переменные используемые для параметров размера
	int size_data = 1; 
	int leng_size_data = 1; //В дальнейшем будет фигурировать эта, как количество строк в сообщение
	int now_size_data = 0;
	int now_size = 0;
	int last_now_size = 0;

	//Мультипликативно выделяем память для массива строк и массива длин строк
	while (elem != EOF) {
		if (size_data == now_size) {
			size_data *= 2;
			all_data = (char*)realloc(all_data, size_data * sizeof(char));
		}
		all_data[now_size++] = elem;

		if (elem == '\n') {
			if (leng_size_data == now_size_data) {
				leng_size_data *= 2;
				data_size = (int*)realloc(data_size, leng_size_data * sizeof(int));
			}

			data_size[now_size_data] = now_size - last_now_size;
			last_now_size = now_size;

			now_size_data++;
		}

		elem = fgetc(file);
	}
	leng_size_data = now_size_data;
	size_data = now_size;
	all_data[size_data] = 0;

	for (int i = 0; i < leng_size_data; i++) {
		all_sum += data_size[i]-1;
	}
	//Инициализируем значения GPU
	char* str; //Сообщение
	int* size_str; //Размер сообщения
	int* tired; //Усталость
	int* stat_way; //Статистика прочитанных символов по каждой строке 
	int* stat_call; //Статистика проходов в сообщение
	//Тут сохраним результаты статистики на CPU
	int* res_stat_way = (int*)malloc(sizeof(int)*leng_size_data); 
	int* res_stat_call = (int*)malloc(sizeof(int)*leng_size_data);
	

	//Выделяем память для на видеокарте
	hipMalloc((void**)&str, sizeof(char) * size_data);
	hipMalloc((void**)&size_str, sizeof(int) * leng_size_data);
	hipMalloc((void**)&tired, sizeof(int));
	hipMalloc((void**)&stat_way, sizeof(int) * leng_size_data);
	hipMalloc((void**)&stat_call, sizeof(int) * leng_size_data);
	
	//Запишем на GPU
	hipMemcpy(str, all_data, sizeof(char) * size_data, hipMemcpyHostToDevice);
	hipMemcpy(size_str, data_size, sizeof(int) * leng_size_data, hipMemcpyHostToDevice);
	hipMemcpy(tired, &stop_call, sizeof(int), hipMemcpyHostToDevice);
	

	dim3 gridSize = dim3(1, 1, 1);    //Размер используемой сетки
	dim3 blockSize = dim3(leng_size_data, 1, 1); //Размер используемого блока

	//Выполняем вызов функции ядра
	AllCall <<<gridSize, blockSize >>> (str, size_str, tired, stat_way, stat_call);

	//Инициализируем и создаем переменную синхронизации потоков
	hipEvent_t syncEvent; 
	
	hipEventCreate(&syncEvent);    //Создаем event
	hipEventRecord(syncEvent, 0);  //Записываем event
	hipEventSynchronize(syncEvent);  //Синхронизируем event

	//Выгружаем данные на CPU
	hipMemcpy(res_stat_call, stat_call, sizeof(int)*leng_size_data, hipMemcpyDeviceToHost);
	hipMemcpy(res_stat_way, stat_way, sizeof(int)*leng_size_data, hipMemcpyDeviceToHost);

	for (int i = 0; i < leng_size_data; i++) {
		printf("call: %d |", res_stat_call[i]);
		printf("way: %d\n", res_stat_way[i]);
	}
	
	hipEventDestroy(syncEvent);

	hipFree(str);
	hipFree(size_str);

	//Обрабатываем статистику на GPU
	int* res_analys;
	int* number_leng;

	//Сохраняем результат в CPU
	int* res_array = (int*)malloc(sizeof(int)*leng_size_data);

	hipMalloc((void**)&res_analys, sizeof(int)* leng_size_data);
	hipMalloc((void**)&number_leng, sizeof(int));

	hipMemcpy(number_leng, &leng_size_data, sizeof(int), hipMemcpyHostToDevice);

	Analys <<< gridSize, blockSize >>> (stat_way, stat_call,  res_analys,  tired, number_leng);

	hipEventCreate(&syncEvent);    //Создаем event
	hipEventRecord(syncEvent, 0);  //Записываем event
	hipEventSynchronize(syncEvent);  //Синхронизируем event

	hipMemcpy(res_array, res_analys, sizeof(int)*leng_size_data, hipMemcpyDeviceToHost);

	int res = 0;
	if (stop_call > leng_size_data) {
		res = all_sum;
	}
	else {
		//Находим максимальное количество правильно считанных элементов
		for (int i = 0; i < leng_size_data; i++) {
			if (res_array[i] > res) {
				res = res_array[i];
			}
		}
	}
	//Вычитаем из всех элемнетов, то есть получаем количество неправильно прочитанных
	printf("FINAL RES: %d", all_sum - res);
	
	hipFree(tired);
	hipFree(stat_way);
	hipFree(stat_call);
	hipFree(res_analys);
	hipFree(number_leng);
	return 0;
}
